#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);//获取输入张量points的第一个维度的大小，即点的数量P
  // 使用 torch::full函数创建一个大小为{P}的张量means，将用于存储每个点到其K近邻的平均距离
  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);
  // 调用函数knn，计算每个点到其K近邻的平均距离，并将结果存储在means张量中
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}