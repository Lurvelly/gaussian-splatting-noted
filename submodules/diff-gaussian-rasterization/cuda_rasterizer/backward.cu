#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

__device__ __forceinline__ float sq(float x) { return x * x; }


// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
/**
  * @brief 计算颜色、位置、球谐系数的梯度
  * @param[in] idx			高斯索引
  * @param[in] deg			球谐函数阶数
  * @param[in] max_coeffs	最大球谐系数
  * @param[in] means		高斯位置
  * @param[in] campos		相机位姿
  * @param[in] shs			球鞋系数
  * @param[in] clamped		标识是否颜色被截断
  * @param[out] dL_dcolor	每个颜色通道的梯度
  * @param[out] dL_dmeans	高斯位置梯度
  * @param[out] dL_dshs		球谐系数梯度
  * @retval None
  */
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	// 计算方向向量
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	// 获取当前高斯的球谐系数
	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	// 被截断的颜色梯度为0
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	// 初始化 RGB对于位置的梯度(x,y,z各个方向上的偏导数)
	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	// 获取用于储存当前高斯球谐系数梯度的地址
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB; //链式求导法则
	if (deg > 0)
	{
		// result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];
		float dRGBdsh1 = -SH_C1 * y;	// d(- SH_C1 * y * sh[1])/dsh[1]
		float dRGBdsh2 = SH_C1 * z;		// d(SH_C1 * z * sh[2])/dsh[2]
		float dRGBdsh3 = -SH_C1 * x;	// d(- SH_C1 * x * sh[3])/dsh[3]
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];		// d(- SH_C1 * x * sh[3])/dx
		dRGBdy = -SH_C1 * sh[1];		// d(- SH_C1 * y * sh[1])/dy
		dRGBdz = SH_C1 * sh[2];			// d(SH_C1 * z * sh[2])/dz

		if (deg > 1)
		{

			/**
			*			result = result +
			*			SH_C2[0] * xy * sh[4] +
			*			SH_C2[1] * yz * sh[5] +
			*			SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
			*			SH_C2[3] * xz * sh[7] +
			*			SH_C2[4] * (xx - yy) * sh[8];
			* */
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				/**
				* 				result = result +
				* 				SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
				* 				SH_C3[1] * xy * z * sh[10] +
				* 				SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
				* 				SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
				* 				SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
				* 				SH_C3[5] * z * (xx - yy) * sh[14] +
				* 				SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
				* */
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	// 计算Loss对方向向量的梯度，使用对 RGB 颜色在各方向的导数梯度 (dRGBdx, dRGBdy, dRGBdz)，通过与颜色梯度 dL_dRGB 点乘，得到损失对方向向量的梯度 dL_ddir
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	// 归一化的方向梯度?
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	// loss 对位置的梯度除了这部分计算的还有后面的，这里只计算了一部分
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)
/**
  * @brief 用于处理高斯的3D协方差矩阵到2D协方差矩阵转换以及计算其反向传播所用的梯度
  * @param[in] P				总高斯数量
  * @param[in] means			高斯位置
  * @param[in] radii			高斯半径
  * @param[in] cov3Ds			高斯协方差矩阵
  * @param[in] h_x				相机焦距 fx
  * @param[in] h_y				相机焦距 fy
  * @param[in] tan_fovx			水平视场角的tan
  * @param[in] tan_fovy			垂直视场角的tan
  * @param[in] view_matrix		视图矩阵(世界->相机)
  * @param[in] opacities		不透明度
  * @param[out] dL_dconics		loss对2D逆协方差矩阵的梯度
  * @param[out] dL_dopacity		loss对不透明度的梯度
  * @param[out] dL_dinvdepth	loss对逆深度的梯度
  * @param[out] dL_dmeans		loss对高斯位置的梯度
  * @param[out] dL_dcov			loss对3D协方差矩阵的梯度
  * @param[in] antialiasing		是否抗锯齿
  * @retval None
  */
__global__ void computeCov2DCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float h_x, float h_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float* opacities,
	const float* dL_dconics,
	float* dL_dopacity,
	const float* dL_dinvdepth,
	float3* dL_dmeans,
	float* dL_dcov,
	bool antialiasing)
{
	// 判断索引是否超出最高斯索引
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// 读取当前高斯对应的3D协方差矩阵(因为对程性，只存储了右上三角，所以只有6个元素)
	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean = means[idx];
	// 这里dL_dconic 初始化为对应高斯的 conic (D2逆协方差矩阵)
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };
	// 从世界坐标到相机坐标
	float3 t = transformPoint4x3(mean, view_matrix);

	// 处理投影边界
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;	//透视除法，得到标准化x坐标
	const float tytz = t.y / t.z;	//透视除法，得到标准化y坐标
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// 用于限制梯度传递是否有效，如果投影在边界之外，梯度值为 0
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

	glm::mat3 J = glm::mat3(h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z),
		0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 T = W * J;

	// 将三维协方差矩阵转换为二维协方差矩阵  Jt*Wt*V*W*J  formula (5)
	glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Use helper variables for 2D covariance entries. More compact.
	float c_xx = cov2D[0][0];
	float c_xy = cov2D[0][1];
	float c_yy = cov2D[1][1];

	constexpr float h_var = 0.3f;	// 对协方差矩阵的微扰动
	float d_inside_root = 0.f;
	if(antialiasing)
	{
		// 抗锯齿情况下，需要对 2D 协方差进行调整，以考虑视角缩放带来的影响。
		const float det_cov = c_xx * c_yy - c_xy * c_xy;
		c_xx += h_var;
		c_yy += h_var;
		const float det_cov_plus_h_cov = c_xx * c_yy - c_xy * c_xy;
		const float h_convolution_scaling = sqrt(max(0.000025f, det_cov / det_cov_plus_h_cov)); // max for numerical stability
		const float dL_dopacity_v = dL_dopacity[idx];
		const float d_h_convolution_scaling = dL_dopacity_v * opacities[idx];
		dL_dopacity[idx] = dL_dopacity_v * h_convolution_scaling;
		d_inside_root = (det_cov / det_cov_plus_h_cov) <= 0.000025f ? 0.f : d_h_convolution_scaling / (2 * h_convolution_scaling);
	} 
	else
	{
		c_xx += h_var;
		c_yy += h_var;
	}
	
	float dL_dc_xx = 0;
	float dL_dc_xy = 0;
	float dL_dc_yy = 0;
	if(antialiasing)
	{
		// https://www.wolframalpha.com/input?i=d+%28%28x*y+-+z%5E2%29%2F%28%28x%2Bw%29*%28y%2Bw%29+-+z%5E2%29%29+%2Fdx
		// https://www.wolframalpha.com/input?i=d+%28%28x*y+-+z%5E2%29%2F%28%28x%2Bw%29*%28y%2Bw%29+-+z%5E2%29%29+%2Fdz
		const float x = c_xx;
		const float y = c_yy;
		const float z = c_xy;
		const float w = h_var;
		const float denom_f = d_inside_root / sq(w * w + w * (x + y) + x * y - z * z);
		// d(det_cov / det_cov_plus_h_cov)/dx = w(wy + y^2 + z^2) / (w^2+w(x+y)+xy-z^2)^2
		const float dL_dx = w * (w * y + y * y + z * z) * denom_f;
		// d(det_cov / det_cov_plus_h_cov)/dy = w(wy + x^2 + z^2) / (w^2+w(x+y)+xy-z^2)^2
		const float dL_dy = w * (w * x + x * x + z * z) * denom_f;
		// d(det_cov / det_cov_plus_h_cov)/dz = 2wz(w + x + y) / (w^2+w(x+y)+xy-z^2)^2
		const float dL_dz = -2.f * w * z * (w + x + y) * denom_f;
		dL_dc_xx = dL_dx;
		dL_dc_yy = dL_dy;
		dL_dc_xy = dL_dz;
	}
	
	float denom = c_xx * c_yy - c_xy * c_xy;

	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

	if (denom2inv != 0)
	{
		// 利用从 2D 协方差矩阵分量计算得到的梯度，计算每个 3D 协方差矩阵分量的梯度
		// Gradients of loss w.r.t. entries of 2D covariance matrix,
		// given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
		// e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
		
		dL_dc_xx += denom2inv * (-c_yy * c_yy * dL_dconic.x + 2 * c_xy * c_yy * dL_dconic.y + (denom - c_xx * c_yy) * dL_dconic.z);
		dL_dc_yy += denom2inv * (-c_xx * c_xx * dL_dconic.z + 2 * c_xx * c_xy * dL_dconic.y + (denom - c_xx * c_yy) * dL_dconic.x);
		dL_dc_xy += denom2inv * 2 * (c_xy * c_yy * dL_dconic.x - (denom + 2 * c_xy * c_xy) * dL_dconic.y + c_xx * c_xy * dL_dconic.z);
		
		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry,
		// given gradients w.r.t. 2D covariance matrix (diagonal).
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 0] = (T[0][0] * T[0][0] * dL_dc_xx + T[0][0] * T[1][0] * dL_dc_xy + T[1][0] * T[1][0] * dL_dc_yy);
		dL_dcov[6 * idx + 3] = (T[0][1] * T[0][1] * dL_dc_xx + T[0][1] * T[1][1] * dL_dc_xy + T[1][1] * T[1][1] * dL_dc_yy);
		dL_dcov[6 * idx + 5] = (T[0][2] * T[0][2] * dL_dc_xx + T[0][2] * T[1][2] * dL_dc_xy + T[1][2] * T[1][2] * dL_dc_yy);
		
		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry,
		// given gradients w.r.t. 2D covariance matrix (off-diagonal).
		// Off-diagonal elements appear twice --> double the gradient.
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 1] = 2 * T[0][0] * T[0][1] * dL_dc_xx + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_dc_xy + 2 * T[1][0] * T[1][1] * dL_dc_yy;
		dL_dcov[6 * idx + 2] = 2 * T[0][0] * T[0][2] * dL_dc_xx + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_dc_xy + 2 * T[1][0] * T[1][2] * dL_dc_yy;
		dL_dcov[6 * idx + 4] = 2 * T[0][2] * T[0][1] * dL_dc_xx + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_dc_xy + 2 * T[1][1] * T[1][2] * dL_dc_yy;
	}
	else
	{
		for (int i = 0; i < 6; i++)
			dL_dcov[6 * idx + i] = 0;
	}

	// Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
	// cov2D = transpose(T) * transpose(Vrk) * T;
	float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_dc_xx +
	(T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc_xy;
	float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_dc_xx +
	(T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc_xy;
	float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_dc_xx +
	(T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc_xy;
	float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc_yy +
	(T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_dc_xy;
	float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc_yy +
	(T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_dc_xy;
	float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc_yy +
	(T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_dc_xy;

	// 梯度传播： T -> J . dL_dJ = dL/dT * dT/dJ
	// Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
	// T = W * J
	float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
	float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
	float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
	float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

	float tz = 1.f / t.z;	//一次倒数
	float tz2 = tz * tz;	//二次倒数
	float tz3 = tz2 * tz;	//三次倒数

	// Gradients of loss w.r.t. transformed Gaussian mean t
	float dL_dtx = x_grad_mul * -h_x * tz2 * dL_dJ02;
	float dL_dty = y_grad_mul * -h_y * tz2 * dL_dJ12;
	float dL_dtz = -h_x * tz2 * dL_dJ00 - h_y * tz2 * dL_dJ11 + (2 * h_x * t.x) * tz3 * dL_dJ02 + (2 * h_y * t.y) * tz3 * dL_dJ12;
	// Account for inverse depth gradients
	if (dL_dinvdepth)
	dL_dtz -= dL_dinvdepth[idx] / (t.z * t.z); //如果存在深度信息，还需要将深度梯度传播到 t.z 的方向上


	// Account for transformation of mean to t
	// t = transformPoint4x3(mean, view_matrix);
	float3 dL_dmean = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the covariance matrix.
	// Additional mean gradient is accumulated in BACKWARD::preprocess.
	dL_dmeans[idx] = dL_dmean;
}

// Backward pass for the conversion of scale and rotation to a 
// 3D covariance matrix for each Gaussian.
/**
  * @brief 计算 3D 协方差矩阵对尺度和旋转参数的梯度
  * @param[in]	idx			高斯索引
  * @param[in]	scale		高斯在x,y,z方向上的缩放系数
  * @param[in]	mod			缩放因子修正系数
  * @param[in]	rot			高斯相对有单位I阵的旋转(四元数w,x,y,z)
  * @param[out] dL_dcov3Ds	loss对协方差矩阵的梯度
  * @param[out] dL_dscales	loss对缩放因子的梯度
  * @param[out] dL_drots	loss对旋转四元数的梯度
  * @retval None
  */
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds, glm::vec3* dL_dscales, glm::vec4* dL_drots)
{
	// 初始化旋转矩阵
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	// 初始化缩放矩阵
	glm::vec3 s = mod * scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	// 缩放*旋转=仿射变换矩阵M
	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	// dunc/ounc 未使用
	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	// 转换为协方差矩阵的形式
	// 在梯度反向传播中，由于非对角线元素对称性带来的重复性，因此需要对这些元素的梯度乘以 0.5 来进行校正
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// 计算缩放因子梯度
	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// 计算对旋转的梯度
	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	float4* dL_drot = (float4*)(dL_drots + idx);
	*dL_drot = float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };//dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
/**
  * @brief
  * @param[in] P				高斯数量
  * @param[in] D				球谐函数阶数
  * @param[in] M				最大球谐系数
  * @param[in] means			高斯的位置
  * @param[in] radii			高斯的半径
  * @param[in] shs				球谐系数
  * @param[in] clamped			用于记录颜色截断情况
  * @param[in] scales			缩放系数
  * @param[in] rotations		表示旋转量的四元素
  * @param[in] scale_modifier	缩放修正系数
  * @param[in] proj				投影矩阵，用于将 3D 坐标转换为 2D 屏幕坐标
  * @param[in] campos			相机位置
  * @param[out] dL_dmean2D		loss对2D坐标的梯度
  * @param[out] dL_dmeans		loss对3D坐标的梯度
  * @param[out] dL_dcolor		loss对颜色的梯度
  * @param[out] dL_dcov3D		loss对3D协方差矩阵的梯度
  * @param[out] dL_dsh			loss对球谐系数的梯度
  * @param[out] dL_drot			loss对旋转的梯度
  * @param[out] dL_dopacity		透明度的梯度，程序中未计算
  * @retval None
  */
template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* proj,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	glm::vec3* dL_dmeans,
	float* dL_dcolor,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot,
	float* dL_dopacity)
{
	// 获取当前线程的索引，使用 cooperative_groups 来计算。
	auto idx = cg::this_grid().thread_rank();

	// 判断是否半径小于等于0，不处理半径小于等于0的高斯
	if (idx >= P || !(radii[idx] > 0))
		return;

	// 获取高斯位置
	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	float4 m_hom = transformPoint4x4(m, proj); // 将高斯位置通过投影矩阵进行投影。
	float m_w = 1.0f / (m_hom.w + 0.0000001f); // 透视除法 + 0.0000001f 防止除0

	// 根据2D的梯度计算3D的梯度
	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean;
	float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]) * m_w * m_w;
	float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]) * m_w * m_w;
	dL_dmean.x = (proj[0] * m_w - proj[3] * mul1) * dL_dmean2D[idx].x + (proj[1] * m_w - proj[3] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.y = (proj[4] * m_w - proj[7] * mul1) * dL_dmean2D[idx].x + (proj[5] * m_w - proj[7] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.z = (proj[8] * m_w - proj[11] * mul1) * dL_dmean2D[idx].x + (proj[9] * m_w - proj[11] * mul2) * dL_dmean2D[idx].y;

	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	// 累加对应高斯的梯度  （computeColorFromSH)部分也会计算这个值，需要累加起来
	dL_dmeans[idx] += dL_dmean;

	// 计算球谐系数的相关梯度
	// Compute gradient updates due to computing colors from SHs
	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3*)means, *campos, shs, clamped, (glm::vec3*)dL_dcolor, (glm::vec3*)dL_dmeans, (glm::vec3*)dL_dsh);

	// 计算3D协方差矩阵相关梯度
	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

// Backward version of the rendering procedure.
/**
  * @brief
  * @param[in]	ranges				每个线程块处理点的范围
  * @param[in]	point_list			高斯点集
  * @param[in]	W					宽度
  * @param[in]	H					高度
  * @param[in]	bg_color			背景颜色
  * @param[in]	points_xy_image		高斯在图像中的位置
  * @param[in]	conic_opacity		2D协方差矩阵的逆与不透明度
  * @param[in]	colors				高斯的颜色
  * @param[in]	depths				高斯的深度
  * @param[out] final_Ts			最终透射率
  * @param[out] n_contrib			每个像素有多少个有贡献的高斯
  * @param[out] dL_dpixels			loss对像素的梯度
  * @param[out] dL_invdepths
  * @param[out] dL_dmean2D			loss对2D坐标的梯度
  * @param[out] dL_dconic2D			loss对2D逆协方差矩阵的梯度
  * @param[out] dL_dopacity			loss对不透明度的梯度
  * @param[out] dL_dcolors			loss对颜色的梯度
  * @param[out] dL_dinvdepths		loss对逆深度的梯度
  * @retval None
  */
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_invdepths,
	float3* __restrict__ dL_dmean2D,
	float4* __restrict__ dL_dconic2D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors,
	float* __restrict__ dL_dinvdepths
)
{
	// We rasterize again. Compute necessary block info.
	// 计算当前CUDA块和线程块对应的像素位置
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	// 检测像素是否在图像范围(W,H)内
	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	// 声明共享内存，用来来存储批量加载的高斯数据
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_depths[BLOCK_SIZE];


	// 初始化一些变量
	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];
	float dL_invdepth;
	float accum_invdepth_rec = 0;
	if (inside)
	{
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
		if(dL_invdepths)
		dL_invdepth = dL_invdepths[pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };
	float last_invdepth = 0;


	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// 分批次处理高斯
	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];

			if(dL_invdepths)
			collected_depths[block.thread_rank()] = depths[coll_id];
		}
		block.sync(); //确保共享内存中数据加载完毕后再进行计算

		// 遍历高斯
		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// 这里主要是计算高斯对当前像素贡献
			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];
			const float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// 计算当前像素的高斯权重 G，G = exp(power)
			const float G = exp(power);
			// 计算 alpha，即当前高斯点对像素的不透明度贡献，保证 alpha 不超过 0.99 以避免完全遮挡。
			const float alpha = min(0.99f, con_o.w * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T; // 计算dchannel/dcolor?

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			// 遍历每个颜色通道
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c; //更新最近使用的颜色

				// 获取loss相对于像素颜色的梯度
				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				// 通过原子加法确保线程安全
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}
			// Propagate gradients from inverse depth to alphaas and
			// per Gaussian inverse depths
			//计算逆深度的梯度
			if (dL_dinvdepths)
			{
			// 当前高斯的逆深度
			const float invd = 1.f / collected_depths[j];
			// 更新累积逆深度贡献
			accum_invdepth_rec = last_alpha * last_invdepth + (1.f - last_alpha) * accum_invdepth_rec;
			// 更新最近使用的深度
			last_invdepth = invd;
			// 更新loss对alpha的梯度，累加与逆深度相关部分
			dL_dalpha += (invd - accum_invdepth_rec) * dL_invdepth;
			atomicAdd(&(dL_dinvdepths[global_id]), dchannel_dcolor * dL_invdepth);
			}

			// 更新loss对alpha的梯度 (dL/dalpha = T * dL/dalpha)
			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			// 计算loss与高斯权重的梯度
			const float dL_dG = con_o.w * dL_dalpha;
			// 计算高斯权重对像素位置的偏微分
			const float gdx = G * d.x;	// dG = G * dx
			const float gdy = G * d.y;  // dG = G * dy
			const float dG_ddelx = -gdx * con_o.x - gdy * con_o.y;
			const float dG_ddely = -gdy * con_o.z - gdx * con_o.y;

			// 更新loss相对于高斯点2D位置的梯度
			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx * ddelx_dx);
			atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely * ddely_dy);

			// 更新loss相对于2D协方差的梯度
			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconic2D[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].y, -0.5f * gdx * d.y * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].w, -0.5f * gdy * d.y * dL_dG);

			// 更新loss想对于不透明度的梯度
			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}

void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const float* opacities,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3* campos,
	const float3* dL_dmean2D,
	const float* dL_dconic,
	const float* dL_dinvdepth,
	float* dL_dopacity,
	glm::vec3* dL_dmean3D,
	float* dL_dcolor,
	float* dL_dcov3D,
	float* dL_dsh,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot,
	bool antialiasing)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		opacities,
		dL_dconic,
		dL_dopacity,
		dL_dinvdepth,
		(float3*)dL_dmean3D,
		dL_dcov3D,
		antialiasing);

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS> << < (P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		radii,
		shs,
		clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		projmatrix,
		campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		dL_dscale,
		dL_drot,
		dL_dopacity);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* bg_color,
	const float2* means2D,
	const float4* conic_opacity,
	const float* colors,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_invdepths,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dcolors,
	float* dL_dinvdepths)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		bg_color,
		means2D,
		conic_opacity,
		colors,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_invdepths,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dcolors,
		dL_dinvdepths
		);
}
